/*

bfield.c 

Computes B fields in toroidal coordinates with given coefficients 
and calculates rms deviation from data

Written by Hee Sok Chung at ANL
July 10, 2016

*/


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include <TTree.h>
//#include <TFile.h>
#include <iostream>
#include <hip/hip_runtime.h>

/* hypergeometric function 1F2 */
double h1f2(double a, double b, double c, double z);

/* LegendreQ(m-1/2,n,cosh(x)) */
double LegendreQ(double m, double n, double z);

/* derivative of LegendreQ(m-1/2,n,cosh(x)) */
double DLegendreQ(int m, int n, double z);

/* coordinate transformation r, z -> zeta */
double zetaf(double rho, double z, double r0);

/* coordinate transformation r, z -> eta */
double etaf(double rho, double z, double r0);

typedef struct B_Struct{
  double Prb[25];
}B_Struct;

using namespace std;

__global__ void thread_call(const double *cc,const double *cs,const double *sc,const double *ss,const double *zeta,const double *eta,const double *Tlegq,const double *Tdleq,const int *midx,const int *nidx,const int*ccidx,const double *dataread,double *dataoutZ,double *dataoutR,double *dataoutPhi,const double rr,const double bzero,const int dim1,int nphi);

int main () {
  double *cc, *cs, *sc, *ss, *lsigma;
  int *ccidx;
  int *midx;
  int *nidx;
  double *Tlegq;
  double *Tdleq;
  double *zeta, *eta;
  double * dataread;
  double * dataoutZ;
  double * dataoutR;
  double * dataoutPhi;
  hipError_t err = hipSuccess;

  zeta = (double*)malloc(30*sizeof(double));
  eta = (double*)malloc(30*sizeof(double));

    int nmax, mmax, nphi, dim1, dim2;
    FILE *input, *iname, *datafile, *datac , *output;
    int Nthread = 8;
//    input = fopen ("fitcoeffN.txt", "r");
    input = fopen ("fitc.txt", "r");
    iname = fopen ("outnames.txt", "r");
    output = fopen ("output52full.txt","w");
    fscanf (iname, "%d", &mmax); // read m Maximum
    fscanf (iname, "%d", &nmax); // read n Maximum
//    fscanf (input, "%d", &mmax); // read m Maximum
//    fscanf (input, "%d", &nmax); // read n Maximum
    printf("MMax %d, NMax %d \n",mmax, nmax);
    dim1=(nmax+1)*(mmax+1); // coefficients dimensions 
    double cterm, rr;
//    int ccidx[mmax+2][nmax+2];
    double zt, et, zt0, bzero, br, bz, bphi, bsize;
    double difrms, dzrms, ffrms, rmsd;
    double angles[30], radii[30], bfield[30][5];
    int i, j, id, idx, md, nd, mdx, ndx, prb, na, ma, qidx, vid;
    double datatmp, tp, tx, phi, wgt, iwgt, legq, dleq;
    double phidz, phidp, phide, phide1, phide2;
    B_Struct B_Measure;
    B_Struct B_Fit;
    B_Struct B_ZFit;
    B_Struct B_RFit;
    B_Struct B_PhiFit;
    cc=(double *)malloc(sizeof*cc*(dim1+2));
    cs=(double *)malloc(sizeof*cs*(dim1+2));
    sc=(double *)malloc(sizeof*sc*(dim1+2));
    ss=(double *)malloc(sizeof*ss*(dim1+2));
    lsigma=(double *)malloc(sizeof*lsigma*(nmax+2));
    int ccidx_dim = dim1+2;
    ccidx=(int *)malloc((dim1+2)*(dim1+2)*sizeof(int));
    if(ccidx==NULL){
        printf("out of memory\n");
    }
    midx=(int *)malloc((dim1+2)*sizeof(int*));
    nidx=(int *)malloc((dim1+2)*sizeof(int*));


    bzero=61.789; // average B-field 
    zt0=6.5;          // eigenfunction normalization point
//  construct probe positions 
    angles[1]=0;
    radii[1]=0;
    for (i=2;i<=9;i++) {
        angles[i]=(i-2.)*M_PI/4;
        radii[i]=22.5;
    }
    for (i=10;i<=25;i++) {
        angles[i]=(i-10.)*M_PI/8;
        radii[i]=45.;
    }
    for (i=1;i<=25;i++) {
        printf("probe %d location r=%.17g, theta=%.17g \n",i,radii[i],angles[i]);
    }

//  calculate toroidal coordinates 
    rr=7111.5; // toroid center 
    printf ("\nComputing coordinates\n");
    for (i=1;i<=25;i++) {
        zeta[i]=zetaf(radii[i]*sin(angles[i])+7112., \
                      -radii[i]*cos(angles[i]),rr);
        eta[i]=etaf(radii[i]*sin(angles[i])+7112., \
                    -radii[i]*cos(angles[i]),rr);
    }
    /* initialize arrays */
    na=0;
    ma=0;
    i=0;
    nd=0;
    md=0;
    lsigma[0]=1.;
    lsigma[nmax]=0.;
    for (nd=0;nd<=nmax;nd++) {
        for (md=0;md<=mmax;md++) {
            i+=1;
            ccidx[md*ccidx_dim+nd]=i;
            midx[i]=md;
            nidx[i]=nd;
            cc[i]=0.; cs[i]=0.; sc[i]=0.; ss[i]=0.;
        }
        if(nd>0&&nd<nmax) {
            lsigma[nd]=sin(M_PI*(nd*1.)/(nmax*1.));
            lsigma[nd]=lsigma[nd]/(M_PI*(nd*1.)/(nmax*1.));
        }

    }
    for (idx=1;idx<=dim1*4;idx++) { // read coefficients
        fscanf (iname, "%d \t %d \t %d", &id, &ma, &na); 
//        fscanf (input, "%d \t %d \t %d \t %lg", &id, &ma, &na, &tp); 
        fscanf (input, "%lg", &tp); 
//        printf("%d, %d, %lg \n", id, ccidx[ma*ccidx_dim+na], tp);
//        if (tp!=tp) printf("%d, %d, %lg \n", id, ccidx[ma][na], tp);
        if (id==1) {
//            printf("cc \n");
            cc[ccidx[ma*ccidx_dim+na]]=tp;
        }
        else if (id==2) {
//            printf("cs \n");
            cs[ccidx[ma*ccidx_dim+na]]=tp;
        }
        else if (id==3) {
//            printf("sc \n");
            sc[ccidx[ma*ccidx_dim+na]]=tp;
        }
        else if (id==4) {
//            printf("ss \n");
            ss[ccidx[ma*ccidx_dim+na]]=tp;
        }
        else {
            printf("error \n");
        }
//        printf("%.20lg \n",tp);
    }
    fclose(input); // close data 
    fclose(iname); // close data 
    printf ("\nComputing b-fields\n");
    nphi=0;
    datac=fopen("data52.txt", "r"); // open data for counting
    while(fscanf(datac, "%lg", &datatmp)>0) {
    nphi++;
    }
    nphi=nphi/26; // azimuthal angle + 25 probes 
    fclose(datac); // close data file used for counting 

    //Loading data first
    dataread = (double *)malloc(sizeof(double)*26*nphi);
    dataoutZ = (double *)malloc(sizeof(double)*26*nphi);
    dataoutR = (double *)malloc(sizeof(double)*26*nphi);
    dataoutPhi = (double *)malloc(sizeof(double)*26*nphi);
    double tmp;
    datafile = fopen ("data52.txt", "r");
    for (id=0;id<nphi;id++) { // loop over azimuthal slices 
      fscanf (datafile, "%lg", &tmp); // read azimuthal angle in degrees 
      dataread[id*26]=tmp*M_PI/180.;
      for (prb=1;prb<=25;prb++){ // loop over 25 probes 
	fscanf (datafile, "%lg", &tmp); 
	dataread[id*26+prb]=tmp*0.001+61.7400000;
      }
    }
    fclose(datafile);
    //Zero output
    for (id=0;id<nphi;id++) { // loop over azimuthal slices 
      for (prb=1;prb<=25;prb++){ // loop over 25 probes 
	dataoutZ[id*26+prb]=0.0;
	dataoutR[id*26+prb]=0.0;
	dataoutPhi[id*26+prb]=0.0;
      }
    }
    //Calculate legq and dleq table
    Tlegq=(double *)malloc((dim1+1)*26*sizeof(double));
    Tdleq=(double *)malloc((dim1+1)*26*sizeof(double));

    for (prb=0;prb<=25;prb++){ // loop over 25 probes 
      for (i=0;i<=dim1;i++){
	Tlegq[i*26+prb]=0.0;
	Tdleq[i*26+prb]=0.0;
      }
    }
    for (prb=1;prb<=25;prb++){ // loop over 25 probes 
      for (i=1;i<=dim1;i++){
	int md=midx[i]; // m index 
	int nd=nidx[i]; // n index 
	/* LegendreQ at probe */
	Tlegq[i*26+prb]=LegendreQ(md,nd,zeta[prb])/LegendreQ(md,nd,zt0);
	/* Derivative of LegendreQ at probe */
	Tdleq[i*26+prb]=DLegendreQ(md,nd,zeta[prb])/LegendreQ(md,nd,zt0);
      }
    }

    //
    difrms=0.;
    dzrms=0.;
    ffrms=0.;

    //Allocate memory in device
    //Function Talbe
    double *d_legq = NULL;
    double *d_dleq = NULL;
    int f_size = sizeof(double)*26*(dim1+1);
    err = hipMalloc((void **)&d_legq, f_size);
    err = hipMalloc((void **)&d_dleq, f_size);

    //Cooridinates
    double *d_zeta = NULL;
    double *d_eta = NULL;
    int c_size = sizeof(double)*30;
    err = hipMalloc((void **)&d_zeta, c_size);
    err = hipMalloc((void **)&d_eta, c_size);

    //data storage;
    double *d_Data = NULL;
    double *d_DataoutZ = NULL;
    double *d_DataoutR = NULL;
    double *d_DataoutPhi = NULL;
    size_t sizeData = sizeof(double)*26*nphi;
    err = hipMalloc((void **)&d_Data, sizeData);
    err = hipMalloc((void **)&d_DataoutZ, sizeData);
    err = hipMalloc((void **)&d_DataoutR, sizeData);
    err = hipMalloc((void **)&d_DataoutPhi, sizeData);

    //ccidx
    int *d_ccidx = NULL;
    int size_ccidx = (dim1+2)*(dim1+2)*sizeof(int);
    err = hipMalloc((void **)&d_ccidx, size_ccidx);

    int *d_midx = NULL;
    int size_midx = (dim1+2)*sizeof(int);
    err = hipMalloc((void **)&d_midx, size_midx);

    int *d_nidx = NULL;
    int size_nidx = (dim1+2)*sizeof(int);
    err = hipMalloc((void **)&d_nidx, size_nidx);

    //Vectors
    double *d_cc = NULL;
    double *d_cs = NULL;
    double *d_sc = NULL;
    double *d_ss = NULL;
    int d_size = sizeof(double)*(dim1+2);
    err = hipMalloc((void **)&d_cc, d_size);
    err = hipMalloc((void **)&d_cs, d_size);
    err = hipMalloc((void **)&d_sc, d_size);
    err = hipMalloc((void **)&d_ss, d_size);

    if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }


    //copy matrix memory to device
    err = hipMemcpy(d_legq, Tlegq, f_size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_dleq, Tdleq, f_size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_zeta, zeta, c_size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_eta, eta, c_size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_Data, dataread, sizeData, hipMemcpyHostToDevice);
    err = hipMemcpy(d_DataoutZ, dataoutZ, sizeData, hipMemcpyHostToDevice);
    err = hipMemcpy(d_DataoutR, dataoutR, sizeData, hipMemcpyHostToDevice);
    err = hipMemcpy(d_DataoutPhi, dataoutPhi, sizeData, hipMemcpyHostToDevice);
    err = hipMemcpy(d_ccidx, ccidx, size_ccidx, hipMemcpyHostToDevice);
    err = hipMemcpy(d_midx, midx, size_midx, hipMemcpyHostToDevice);
    err = hipMemcpy(d_nidx, nidx, size_nidx, hipMemcpyHostToDevice);
    err = hipMemcpy(d_cc, cc, d_size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_cs, cs, d_size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_sc, sc, d_size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_ss, ss, d_size, hipMemcpyHostToDevice);



    //Start parallel computing
    dim3 DimBlock (16,16);
    dim3 DimGrid (nphi/16+1, 26/16+1);
    printf("CUDA kernel launch with %d blocks of %d threads\n", DimGrid.x * DimGrid.y, 256);

    thread_call<<<DimGrid, DimBlock>>>(d_cc,d_cs,d_sc,d_ss,d_zeta,d_eta,d_legq,d_dleq,d_midx,d_nidx,d_ccidx,d_Data,d_DataoutZ,d_DataoutR,d_DataoutPhi,rr,bzero,dim1,nphi);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    printf("GPU jobs done.\n");


    //End parallel computing
    err = hipMemcpy(dataoutZ, d_DataoutZ, sizeData, hipMemcpyDeviceToHost);
    err = hipMemcpy(dataoutR, d_DataoutR, sizeData, hipMemcpyDeviceToHost);
    err = hipMemcpy(dataoutPhi, d_DataoutPhi, sizeData, hipMemcpyDeviceToHost);
    
    //open Root tree
/*    TFile *outfile = new TFile("RootOut52.root","recreate");
    TTree *Tree_Measured = new TTree ("Tree_Measured", "Measured field");
    Tree_Measured->Branch("Phi",&phi,"Phi/D");
    Tree_Measured->Branch("BField",&B_Measure,"Prb1/D:Prb2:Prb3:Prb4:Prb5:Prb6:Prb7:Prb8:Prb9:Prb10:Prb11:Prb12:Prb13:Prb14:Prb15:Prb16:Prb17:Prb18:Prb19:Prb20:Prb21:Prb22:Prb23:Prb24:Prb25");
    TTree *Tree_Fit = new TTree ("Tree_Fit", "Fitted field");
    Tree_Fit->Branch("Phi",&phi,"Phi/D");
    Tree_Fit->Branch("BField",&B_Fit,"Prb1/D:Prb2:Prb3:Prb4:Prb5:Prb6:Prb7:Prb8:Prb9:Prb10:Prb11:Prb12:Prb13:Prb14:Prb15:Prb16:Prb17:Prb18:Prb19:Prb20:Prb21:Prb22:Prb23:Prb24:Prb25");
    Tree_Fit->Branch("BFieldZ",&B_Fit,"Prb1/D:Prb2:Prb3:Prb4:Prb5:Prb6:Prb7:Prb8:Prb9:Prb10:Prb11:Prb12:Prb13:Prb14:Prb15:Prb16:Prb17:Prb18:Prb19:Prb20:Prb21:Prb22:Prb23:Prb24:Prb25");
    Tree_Fit->Branch("BFieldR",&B_Fit,"Prb1/D:Prb2:Prb3:Prb4:Prb5:Prb6:Prb7:Prb8:Prb9:Prb10:Prb11:Prb12:Prb13:Prb14:Prb15:Prb16:Prb17:Prb18:Prb19:Prb20:Prb21:Prb22:Prb23:Prb24:Prb25");
    Tree_Fit->Branch("BFieldPhi",&B_Fit,"Prb1/D:Prb2:Prb3:Prb4:Prb5:Prb6:Prb7:Prb8:Prb9:Prb10:Prb11:Prb12:Prb13:Prb14:Prb15:Prb16:Prb17:Prb18:Prb19:Prb20:Prb21:Prb22:Prb23:Prb24:Prb25");
    */
    for (id=0;id<nphi;id++) { // loop over azimuthal slices 
      phi=dataread[id*26];
      for (prb=1;prb<=25;prb++){ // loop over 25 probes 
	bz=dataoutZ[id*26+prb];
	br=dataoutR[id*26+prb];
	bphi=dataoutPhi[id*26+prb];
	bsize=sqrt(br*br+bz*bz+bphi*bphi);
	//            printf("%Lf \n",bsize);
	bfield[prb][4]=bsize;
	bfield[prb][1]=bz;
	bfield[prb][2]=br;
	bfield[prb][3]=bphi;
	B_Fit.Prb[prb-1]=bsize;
	B_ZFit.Prb[prb-1]=bz;
	B_RFit.Prb[prb-1]=br;
	B_PhiFit.Prb[prb-1]=bphi;
	B_Measure.Prb[prb-1]=dataread[id*26+prb];
	//Output
	fprintf(output,"%.17g %d %.17g %.17g %.17g %.17g %.17g\n",phi,prb,dataread[id*26+prb],bfield[prb][1],bfield[prb][2],bfield[prb][3],bfield[prb][4]);
	rmsd=(dataread[id*26+prb]-bfield[prb][4])*(dataread[id*26+prb]-bfield[prb][4]);
	difrms+=rmsd;
	rmsd=rmsd/bzero/bzero;
	dzrms+=(dataread[id*26+prb]-bz)*(dataread[id*26+prb]-bz);
	ffrms+=(dataread[id*26+prb]-bzero)*(dataread[id*26+prb]-bzero);
      }
     // Tree_Measured->Fill();
     // Tree_Fit->Fill();
    }
    //Tree_Measured->Write();
    //Tree_Fit->Write();
    //outfile->Close();

    difrms=difrms/bzero/bzero/nphi/25;
    dzrms=dzrms/bzero/bzero/nphi/25;
    ffrms=ffrms/bzero/bzero/nphi/25;
    difrms=sqrt(difrms)*1000000.;
    dzrms=sqrt(dzrms)*1000000.;
    ffrms=sqrt(ffrms)*1000000.;
    printf("RMS fluctuation = %.17g ppm \n", ffrms);
    printf("RMS difference (lin. approx.) = %.17g ppm \n", dzrms);
    printf("RMS difference (real) = %.17g ppm \n", difrms);

    free(cc);
    free(cs);
    free(sc);
    free(ss);
    free(ccidx);
    free(midx);
    free(nidx);
    free(Tlegq);
    free(Tdleq);
    free(dataread);
    free(dataoutZ);
    free(dataoutR);
    free(dataoutPhi);

    return 0;
}

/* hypergeometric function 1F2 
   evaluated by truncating an infinite sum */
double h1f2(double a, double b, double c, double z){
	int i, imax;
	double err, tol, si, sii, f1f2;
	double errabs, errrel;
	imax=100000000; // 10^8 maximum iterations
	sii=1.; // initial term 
	err=1.; // estimated uncertainty 
	f1f2=1.; // initial contribution 
	i=0;  // iterator
	tol=pow(10.,-16.); // error tolerance 
	if (z<tol||z>1.-tol) { // z out of range or dangerously close to 0 or 1
		i=imax+1;
		f1f2=1./err; // to return inf or nan
	}
	while (err>tol&&i<=imax) {
		i++;
		si=(a+i-1.)*(b+i-1.)/(c+i-1.)*z/i*sii; // next term 
		f1f2+=si; //next term added 
		errabs=fabsl(si*z/(1.-z)); // estimated absolute uncertainty 
		errrel=fabsl(errabs/f1f2); // estimated relative uncertainty 
		if (errabs>errrel) { // choose larger one as error
			err=errabs;
		}
		else {
			err=errrel;
		}
		sii=si;
	}
	return f1f2;
}

/* Legendre function of the second kind, Q_{m+1/2}^n (cosh(z)).
   Normalized to remove gamma function.
   Regular inside the torus */
double LegendreQ(double m, double n, double z){
    double lq;
//    lq=pow(M_PI,.5)*exp(lgammal(m+n+.5))/(pow(2.,m+.5)*exp(lgammal(m+1.)));
    lq=pow(tanh(z),n)/pow(cosh(z),m+.5);
    lq=lq*h1f2(.5*(m+n+.5),.5*(m+n+1.5),m+1.,1./cosh(z)/cosh(z));
    return lq;
}

/* Derivative of Q_{m+1/2}^n (cosh(z)).
   Normalized to remove gamma function.
 */
double DLegendreQ(int m, int n, double z){
    double dlq, lq1, lq2;
    if (m==0) {
        dlq=-1/(8*pow(cosh(z),1.5))/sinh(z);
        dlq=dlq*pow(tanh(z),n);
        dlq=dlq*( (4.*pow(sinh(z),2.)-8.*n)*                        \
                   h1f2(n/2.+.25,n/2.+.75,1.,1/cosh(z)/cosh(z)) +   \
                   (4.*n*n+8.*n+3.)*tanh(z)*tanh(z)*                \
                   h1f2(n/2.+1.25,n/2.+1.75,2.,1/cosh(z)/cosh(z)));
    }
    else {
        lq1=pow(tanh(z),n)/pow(cosh(z),m+.5);
        lq1=lq1*h1f2(.5*(m+n+.5),.5*(m+n+1.5),m+1.,1./cosh(z)/cosh(z));
        lq2=pow(tanh(z),n)/pow(cosh(z),m-.5);
        lq2=lq2*h1f2(.5*(m+n-.5),.5*(m+n+.5),m*1.,1./cosh(z)/cosh(z));
        dlq=(m-.5)/tanh(z)*lq1-(2.*m)/sinh(z)*lq2;
    }
    return dlq;
}

/* coordinate transformation r, z -> zeta */
double zetaf(double rho, double z, double r0){
    double zetax;
    zetax=atanhl(2.*rho*r0/(rho*rho+r0*r0+z*z));
    return zetax;
}

/* coordinate transformation r, z -> eta */
double etaf(double rho, double z, double r0){
    double etax, xx;
    int i;
    xx=2.*r0*z/(rho*rho-r0*r0+z*z);
    if (fabsl(xx)<0.001) {
        etax=1.;
        for (i=1;i<=10;i++) {
            etax=etax+pow(xx,2.*i)/(2.*i+1.)*cos(M_PI*i);
        }
        etax=etax*xx;
    }
    else {
        etax=atanl(xx);
    }
    if (rho<sqrtl(r0*r0-z*z)) etax=etax+M_PI;
    return etax;
}


//Function for each thread call
__global__ void thread_call(const double *cc,const double *cs,const double *sc,const double *ss,const double *zeta,const double *eta,const double *Tlegq,const double *Tdleq,const int *midx,const int *nidx,const int*ccidx,const double *dataread,double *dataoutZ,double *dataoutR,double *dataoutPhi,const double rr,const double bzero,const int dim1,int nphi){
  //Get index
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  int prb = blockDim.y * blockIdx.y + threadIdx.y;
  int ccidx_dim = dim1+2;
  if(id<nphi && prb<26 && prb>0){
    double phi=dataread[id*26];
    double zt=zeta[prb];  // zeta coordinate at probe 
    double et=eta[prb];  // eta coordinate at probe 
    //            printf("%Lf, %Lf \n", zt, et);
    double wgt=sqrt(cosh(zt)-cos(et)); // weight func in toroidal coordinates 
    double br=0.; 
    double bz=bzero; 
    double bphi=0.;
    for (int i=1;i<=dim1;i++){
      int md=midx[i]; // m index 
      int nd=nidx[i]; // n index 
      //                printf("%d, %d \n", md, nd);
      /* LegendreQ at probe */
      double legq=Tlegq[i*26+prb];
      /* Derivative of LegendreQ at probe */
      double dleq=Tdleq[i*26+prb];
      //                printf("(%d, %d, %Lf) : %Lf, %Lf \n", md,nd,zt,legq, dleq);
      double phidz=cc[ccidx[md*ccidx_dim+nd]]*cos(nd*phi)*cos(md*et)+  \
		   sc[ccidx[md*ccidx_dim+nd]]*cos(nd*phi)*sin(md*et)+  \
		   cs[ccidx[md*ccidx_dim+nd]]*sin(nd*phi)*cos(md*et)+  \
		   ss[ccidx[md*ccidx_dim+nd]]*sin(nd*phi)*sin(md*et);
      phidz=phidz*(sinh(zt)/2./wgt*legq + wgt*dleq);
      double phidp=-cc[ccidx[md*ccidx_dim+nd]]*sin(nd*phi)*cos(md*et)-  \
		   sc[ccidx[md*ccidx_dim+nd]]*sin(nd*phi)*sin(md*et)+  \
		   cs[ccidx[md*ccidx_dim+nd]]*cos(nd*phi)*cos(md*et)+  \
		   ss[ccidx[md*ccidx_dim+nd]]*cos(nd*phi)*sin(md*et);
      phidp=phidp*nd*wgt*legq;
      double phide1=cc[ccidx[md*ccidx_dim+nd]]*cos(nd*phi)*cos(md*et)+  \
		    sc[ccidx[md*ccidx_dim+nd]]*cos(nd*phi)*sin(md*et)+  \
		    cs[ccidx[md*ccidx_dim+nd]]*sin(nd*phi)*cos(md*et)+  \
		    ss[ccidx[md*ccidx_dim+nd]]*sin(nd*phi)*sin(md*et);
      phide1=phide1*sin(et)/2./wgt*legq;
      double phide2=-cc[ccidx[md*ccidx_dim+nd]]*cos(nd*phi)*sin(md*et)+  \
		    sc[ccidx[md*ccidx_dim+nd]]*cos(nd*phi)*cos(md*et)-  \
		    cs[ccidx[md*ccidx_dim+nd]]*sin(nd*phi)*sin(md*et)+  \
		    ss[ccidx[md*ccidx_dim+nd]]*sin(nd*phi)*cos(md*et);
      phide2=phide2*md*wgt*legq;
      double phide=phide1+phide2;
      br=br+sinh(zt)/rr*((1.-cosh(zt)*cos(et))/sinh(zt)*phidz-sin(et)*phide);
      bz=bz+sinh(zt)/rr*(-sin(et)*phidz-(1.-cosh(zt)*cos(et))/sinh(zt)*phide);
      bphi=bphi+(cosh(zt)-cos(et))/(rr*sinh(zt))*phidp;
    }
    dataoutZ[id*26+prb]=bz;
    dataoutR[id*26+prb]=br;
    dataoutPhi[id*26+prb]=bphi;
  }
}







