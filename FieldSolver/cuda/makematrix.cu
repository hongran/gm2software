/*

makematrix.cu

Constructs matrix equation A x = b for local minimization problem 
Exports matrix A and vector b into plain text

Written by Hee Sok Chung at ANL
July 10, 2016

Modified by Ran Hong for cuda compatibility

 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


/* Legendre functions are defined without normalization factors in order to 
 * avoid overflow from gamma function */

/* hypergeometric function 1F2 */
double h1f2(double a, double b, double c, double z);

/* LegendreQ(m-1/2,n,cosh(x)) */
double LegendreQ(double m, double n, double z);

/* derivative of LegendreQ(m-1/2,n,cosh(x)) */
double DLegendreQ(int m, int n, double z);

/* coordinate transformation r, z -> zeta */
double zetaf(double rho, double z, double r0);

/* coordinate transformation r, z -> eta */
double etaf(double rho, double z, double r0);

/* main subroutine that calculates matrix and vector elements */
void cpx(double **datain, 
    double *cc, double *cs, double *sc, double *ss,
    double **matrix, 
    double *zeta, double *eta, 
    int **ccidx, int ndata, int dim1, 
    double zt0, double rr, int *sliceflag);

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorProd(const double *d_tcc, const double *d_tcs, const double *d_tsc, const double *d_tss, double *d_M, int dim_v)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int ii=0;
  int jj=0;
  int dim_M = 4*dim_v+2;
  double local_M[4][4];


  if (i < dim_v && j< dim_v){
    for (ii=0;ii<4;ii++){
      for (jj=0;jj<4;jj++){
	local_M[ii][jj]=d_M[(4*i+ii+1)*dim_M+4*j+jj+1];
      }
    }

    local_M[0][0]+=d_tcc[i]*d_tcc[j];
    local_M[1][0]+=d_tcs[i]*d_tcc[j];
    local_M[2][0]+=d_tsc[i]*d_tcc[j];
    local_M[3][0]+=d_tss[i]*d_tcc[j]; 

    local_M[0][1]+=d_tcc[i]*d_tcs[j];
    local_M[1][1]+=d_tcs[i]*d_tcs[j];
    local_M[2][1]+=d_tsc[i]*d_tcs[j]; 
    local_M[3][1]+=d_tss[i]*d_tcs[j];

    local_M[0][2]+=d_tcc[i]*d_tsc[j]; 
    local_M[1][2]+=d_tcs[i]*d_tsc[j]; 
    local_M[2][2]+=d_tsc[i]*d_tsc[j]; 
    local_M[3][2]+=d_tss[i]*d_tsc[j]; 

    local_M[0][3]+=d_tcc[i]*d_tss[j]; 
    local_M[1][3]+=d_tcs[i]*d_tss[j]; 
    local_M[2][3]+=d_tsc[i]*d_tss[j]; 
    local_M[3][3]+=d_tss[i]*d_tss[j]; 

    for (ii=0;ii<4;ii++){
      for (jj=0;jj<4;jj++){
	d_M[(4*i+ii+1)*dim_M+4*j+jj+1]=local_M[ii][jj];
      }
    }
  }
}


int main (int argc, char **argv) {
  int nmax, mmax, ndata, dim1, dim2x, vecdim;
  //  Max. number of harmonics in azimuthal direction 
  nmax=200; 
  //  Max. number of harmonics in poloidal direction 
  mmax=8; 
  //  Matrix and vector dimensions 
  dim1=(nmax+1)*(mmax+1);
  dim2x=4*dim1;
  double rr;
  double *cc, *cs, *sc, *ss;
  double **matrix, **datain;
  double datatmp;
  double zeta[30], eta[30];
  double zt0, bzero;
  double angles[30], radii[30];
  int **ccidx;
  int **mnidx;
  int *sinz, *vecidx;
  int i, j, id, md, nd, prb ;
  int vecc;
  FILE *input;
  FILE *inputc;
  //    procnum=11; // number of thread to use 
  //    printf("Using %d threads\n",procnum);

  //  declare vector and matrix elements as pointers 
  cc=(double *)malloc((dim1+2)*sizeof(double));
  cs=(double *)malloc((dim1+2)*sizeof(double));
  sc=(double *)malloc((dim1+2)*sizeof(double));
  ss=(double *)malloc((dim1+2)*sizeof(double));

  matrix=(double **)malloc((dim2x+2)*sizeof(double*));
  for (i=0;i<(dim2x+2);i++) {
    matrix[i]=(double *)malloc((dim2x+2)*sizeof(double));
  }
  sinz=(int *)malloc((dim2x+2)*sizeof(int)); // for identifying exact zeros 
  vecidx=(int *)malloc((dim2x+2)*sizeof(int)); // vector index 
  ccidx=(int **)malloc((dim1+2)*sizeof(int*)); // matrix single index -> double index 
  for (i=0;i<(dim1+2);i++) {
    ccidx[i]=(int *)malloc(3*sizeof(int));
  }
  mnidx=(int **)malloc((dim1+2)*sizeof(int*)); // matrix double index -> single index
  for (i=0;i<(dim1+2);i++) {
    mnidx[i]=(int *)malloc((dim1+2)*sizeof(int));
  }

  //  count number of azimuthal slices 
  ndata=0;
  //  inputc=fopen("data40.txt", "r"); // open data for counting
  inputc=fopen("data52.txt", "r"); // open data for counting
  while(fscanf(inputc, "%lf", &datatmp)>0) {
    ndata++;
  }
  ndata=ndata/26; // azimuthal angle + 25 probes 
  fclose(inputc); // close data file used for counting 
  int sliceflag[ndata+1];
  printf("number of data = %d\n", ndata);
  //  this pointer will be used for data taking 
  datain=(double **)malloc((ndata+1)*sizeof(double*));
  for (id=0;id<=ndata;id++) {
    datain[id]=(double *)malloc((25+2)*sizeof(double));
  }
  //  distribute azimuthal slices to threads
  //    printf("%d\n",procnum);
  //    printf("%d\n",ndatap);
  i=1;
  for (id=1;id<=ndata;id++){
    sliceflag[id]=1;
  }
  //    printf("Max i = %d\n", i);

  bzero=61.789; // reference B field, chosen close to average B field 
  zt0=6.5; // toroidal harmonics will be normalized at zeta=zt0
  //  construct probe positions 
  angles[1]=0;
  radii[1]=0;
  for (i=2;i<=9;i++) {
    angles[i]=(i-2.)*M_PI/4;
    radii[i]=22.5;
  }
  for (i=10;i<=25;i++) {
    angles[i]=(i-10.)*M_PI/8;
    radii[i]=45.;
  }
  /*    for (i=1;i<=25;i++) {
	printf("probe %d location r=%lf, theta=%lf \n",i+1,radii[i],angles[i]);
	} */

  //  calculate toroidal coordinates 
  rr=7111.5; // toroid center. MUST NEVER COINCIDE WITH ACTUAL PROBE POSITION 
  printf ("\nComputing coordinates\n");
  for (i=1;i<=25;i++) {
    zeta[i]=zetaf(radii[i]*sin(angles[i])+7112., \
	-radii[i]*cos(angles[i]),rr);
    eta[i]=etaf(radii[i]*sin(angles[i])+7112., \
	-radii[i]*cos(angles[i]),rr);
  }
  /* initialize arrays */
  i=0;
  nd=0;
  md=0;
  printf ("\nInitializing coefficients\n");
  for (i=0;i<=dim2x;i++) {
    sinz[i]=1;
  }
  i=0;
  for (nd=0;nd<=nmax;nd++) {
    for (md=0;md<=mmax;md++) {
      i+=1;
      ccidx[i][1]=md; ccidx[i][2]=nd; 
      mnidx[md][nd]=4*(i-1);
      cc[i]=0.; 
      cs[i]=0.; 
      sc[i]=0.; 
      ss[i]=0.;
      if(nd==0) { // identify sin(0) = 0 due to n = 0
	sinz[4*(i-1)+2]=0;
	sinz[4*(i-1)+4]=0;
      }
      if(md==0) { // identify sin(0) = 0 due to m = 0
	sinz[4*(i-1)+3]=0;
	sinz[4*(i-1)+4]=0;
      }
    }
  }
  printf ("\nInitializing matrix\n");
  for (i=0;i<=dim2x;i++) {
    for (j=0;j<=dim2x;j++) {
      matrix[i][j]=0.;
    }
  }
  //    input =fopen("data40.txt", "r"); // this one's for actually reading data
  input =fopen("data52.txt", "r"); // this one's for actually reading data
  for (id=1;id<=ndata;id++) {
    fscanf (input, "%lf", &datatmp); // read azimuthal angle in degrees 
    datain[id][0]=datatmp;
    for (prb=1;prb<=25;prb++){ // loop over 25 probes 
      fscanf (input, "%lf", &datatmp); // read B-fields in kHz 
      datatmp=datatmp*0.001+61.7400000-bzero; // convert to MHz and offset
      datain[id][prb]=datatmp;
    }
  }
  fclose(input); // close data 

  ///////////////////////////////////////////////////////////////////////////////////////////////////
  printf ("\nComputing matrix elements\n");
  //  Calculate matrix and vector elements 
  cpx(datain, cc, cs, sc, ss, matrix, 
      zeta, eta, ccidx, ndata, dim1, zt0, rr, sliceflag);

  ///////////////////////////////////////////////////////////////////////////////////////////////////
  for(i=1;i<=ndata;i++) {
    if (sliceflag[i]!=0) printf("slice %d status = %d\n", i, sliceflag[i]);
  }
  //  combine results from individual threads 

  FILE *out0, *out1, *outn, *outd;
  out0 = fopen ("outmatrix.dat", "w"); // matrix elements 
  out1 = fopen ("outvector.dat", "w"); // vector elements 
  outn = fopen ("outnames.txt", "w");  // index dictionary 
  outd = fopen ("vecdim.txt", "w");    // matrix dimensions

  fprintf(outn, "%d\n", mmax);
  fprintf(outn, "%d\n", nmax);

  for (i=1;i<=dim1;i++) {
    fprintf(outn, "1\t%d\t%d\n",ccidx[i][1], ccidx[i][2]);
    if(sinz[4*(i-1)+2]!=0) {
      fprintf(outn, "2\t%d\t%d\n",ccidx[i][1], ccidx[i][2]);
    }
    if(sinz[4*(i-1)+3]!=0) {
      fprintf(outn, "3\t%d\t%d\n",ccidx[i][1], ccidx[i][2]);
    }
    if(sinz[4*(i-1)+4]!=0) {
      fprintf(outn, "4\t%d\t%d\n",ccidx[i][1], ccidx[i][2]);
    }
  }
  vecdim=0;
  for (i=1;i<=dim1;i++) {
    vecdim++;
    //    fprintf(out1, "%.17g\n", cc[i]);
    fwrite(&(cc[i]),sizeof(double),1,out1);
    if(sinz[4*(i-1)+2]!=0) {
      vecdim++;
      //      fprintf(out1, "%.17g\n", cs[i]);
      fwrite(&(cs[i]),sizeof(double),1,out1);
    }
    if(sinz[4*(i-1)+3]!=0) {
      vecdim++;
      //      fprintf(out1, "%.17g\n", sc[i]);
      fwrite(&(sc[i]),sizeof(double),1,out1);
    }
    if(sinz[4*(i-1)+4]!=0) {
      vecdim++;
      //      fprintf(out1, "%.17g\n", ss[i]);
      fwrite(&(ss[i]),sizeof(double),1,out1);
    }
  }
  id=0;
  for (i=1;i<=dim2x;i++) {
    if(sinz[i]!=0) {
      id++;
      vecidx[id]=i;
    }
  }
  vecc=0;
  for (i=1;i<=vecdim;i++) {
    for (j=1;j<=vecdim;j++) {
      vecc++;
//      fprintf(out0, "%.17g\n", matrix[vecidx[i]][vecidx[j]]);
      fwrite(&(matrix[vecidx[i]][vecidx[j]]),sizeof(double),1,out0);
    }
  }
  fprintf(outd, "%d\n", vecdim);

  fclose(out0);
  fclose(out1);
  fclose(outn);
  fclose(outd);
  free(cc);
  free(cs);
  free(sc);
  free(ss);
  free(ccidx);
  return 0;
}

/* hypergeometric function 1F2 
   evaluated by truncating an infinite sum */
double h1f2(double a, double b, double c, double z){
  int i, imax;
  double err, tol, si, sii, f1f2;
  double errabs, errrel;
  imax=100000000; // 10^8 maximum iterations
  sii=1.; // initial term 
  err=1.; // estimated uncertainty 
  f1f2=1.; // initial contribution 
  i=0;  // iterator
  tol=pow(10.,-16.); // error tolerance 
  if (z<tol||z>1.-tol) { // z out of range or dangerously close to 0 or 1
    i=imax+1;
    f1f2=1./i; // to return inf or nan
    i=imax+1; // stop evaluation
  }
  while (err>tol&&i<=imax) {
    i++;
    si=(a+i-1.)*(b+i-1.)/(c+i-1.)*z/i*sii; // next term 
    f1f2+=si; //next term added 
    errabs=fabsl(si*z/(1.-z)); // estimated absolute uncertainty 
    errrel=fabsl(errabs/f1f2); // estimated relative uncertainty 
    if (errabs>errrel) { // choose larger one as error
      err=errabs;
    }
    else {
      err=errrel;
    }
    sii=si;
  }
  return f1f2;
}

/* Legendre function of the second kind, Q_{m+1/2}^n (cosh(z)).
   Normalized to remove gamma function.
   Regular inside the torus */
double LegendreQ(double m, double n, double z){
  double lq;
  lq=pow(tanh(z),n)/pow(cosh(z),m+.5);
  lq=lq*h1f2(.5*(m+n+.5),.5*(m+n+1.5),m+1.,1./cosh(z)/cosh(z));
  return lq;
}

/* Derivative of Q_{m+1/2}^n (cosh(z)).
   Normalized to remove gamma function.
 */
double DLegendreQ(int m, int n, double z){
  double dlq, lq1, lq2;
  if (m==0) {
    dlq=-1/(8*pow(cosh(z),1.5))/sinh(z);
    dlq=dlq*pow(tanh(z),n);
    dlq=dlq*( (4.*pow(sinh(z),2.)-8.*n)*                        \
	h1f2(n/2.+.25,n/2.+.75,1.,1/cosh(z)/cosh(z)) +   \
	(4.*n*n+8.*n+3.)*tanh(z)*tanh(z)*                \
	h1f2(n/2.+1.25,n/2.+1.75,2.,1/cosh(z)/cosh(z)));
  }
  else {
    lq1=pow(tanh(z),n)/pow(cosh(z),m+.5);
    lq1=lq1*h1f2(.5*(m+n+.5),.5*(m+n+1.5),m+1.,1./cosh(z)/cosh(z));
    lq2=pow(tanh(z),n)/pow(cosh(z),m-.5);
    lq2=lq2*h1f2(.5*(m+n-.5),.5*(m+n+.5),m*1.,1./cosh(z)/cosh(z));
    dlq=(m-.5)/tanh(z)*lq1-(2.*m)/sinh(z)*lq2;
  }
  return dlq;
}

/* coordinate transformation r, z -> zeta */
double zetaf(double rho, double z, double r0){
  double zetax;
  zetax=atanh(2.*rho*r0/(rho*rho+r0*r0+z*z));
  return zetax;
}

/* coordinate transformation r, z -> eta */
double etaf(double rho, double z, double r0){
  double etax, xx;
  int i;
  xx=2.*r0*z/(rho*rho-r0*r0+z*z);
  if (fabsl(xx)<0.001) {
    etax=1.;
    for (i=1;i<=10;i++) {
      etax=etax+pow(xx,2.*i)/(2.*i+1.)*cos(M_PI*i);
    }
    etax=etax*xx;
  }
  else {
    etax=atanl(xx);
  }
  if (rho<sqrtl(r0*r0-z*z)) etax=etax+M_PI;
  return etax;
}

void cpx(double **datain, 
    double *cc, double *cs, double *sc, double *ss,
    double **matrix, 
    double *zeta, double *eta, 
    int **ccidx, int ndata, int dim1, 
    double zt0, double rr, int *sliceflag) {
  double datatmp, zt, et, wgt, phi;
  double *tcc, *tcs, *tsc, *tss;
  tcc=(double *)malloc(sizeof*tcc*(dim1));
  tcs=(double *)malloc(sizeof*tcs*(dim1));
  tsc=(double *)malloc(sizeof*tsc*(dim1));
  tss=(double *)malloc(sizeof*tss*(dim1));
  int md, nd, i, id, prb, j ;
  double legq, dleq;
  hipError_t err = hipSuccess;

  //Calculate legq and dleq table
  double **Tlegq;
  double **Tdleq;
  Tlegq=(double **)malloc((dim1+1)*sizeof(double*));
  Tdleq=(double **)malloc((dim1+1)*sizeof(double*));
  for (int il=0;il<=dim1;il++) {
    Tlegq[il]=(double *)malloc(26*sizeof(double));
    Tdleq[il]=(double *)malloc(26*sizeof(double));
  }
  for (prb=1;prb<=25;prb++){ // loop over 25 probes 
    zt=zeta[prb];  // zeta coordinate at probe 
    for (i=1;i<=dim1;i++){
      md=ccidx[i][1]; // m index 
      nd=ccidx[i][2]; // n index 
      /* LegendreQ at probe */
      Tlegq[i][prb]=LegendreQ(md,nd,zeta[prb])/LegendreQ(md,nd,zt0);
      /* Derivative of LegendreQ at probe */
      Tdleq[i][prb]=DLegendreQ(md,nd,zeta[prb])/LegendreQ(md,nd,zt0);
    }
  }
  //Table calculation complete

  //Re-allocate matrix
  double *h_M = NULL;
  int dimM = 4*dim1+2;
  size_t sizeM = dimM*dimM*sizeof(double);
  h_M = (double *)malloc(sizeM);

  //device Malloc
  double *d_tcc = NULL;
  double *d_tcs = NULL;
  double *d_tsc = NULL;
  double *d_tss = NULL;
  int d_size = sizeof(double)*dim1;
  double *d_M = NULL;
  err = hipMalloc((void **)&d_tcc, d_size);
  err = hipMalloc((void **)&d_tcs, d_size);
  err = hipMalloc((void **)&d_tsc, d_size);
  err = hipMalloc((void **)&d_tss, d_size);
  err = hipMalloc((void **)&d_M, sizeM);

  //copy matrix memory to device
  err = hipMemcpy(d_M, h_M, sizeM, hipMemcpyHostToDevice);

  for (id=1;id<=ndata;id++) { // loop over azimuthal slices 
    if (id!=0) { 
      sliceflag[id]=0;
      datatmp=datain[id][0];
      phi=datatmp*M_PI/180.;
      for (prb=1;prb<=25;prb++){ // loop over 25 probes 
	datatmp=datain[id][prb];
	zt=zeta[prb];  // zeta coordinate at probe 
	et=eta[prb];  // eta coordinate at probe 
	wgt=sqrt(cosh(zt)-cos(et)); // weight func in toroidal coordinates 
	//
	double Sinh_zt = sinh(zt);
	double Sin_et = sin(et);
	double Cosh_zt = cosh(zt);
	double Cos_et = cos(et);
	//
	for (i=1;i<=dim1;i++){
	  md=ccidx[i][1]; // m index 
	  nd=ccidx[i][2]; // n index 
	  /* LegendreQ at probe */
	  //legq=LegendreQ(md,nd,zeta[prb])/LegendreQ(md,nd,zt0);
	  legq=Tlegq[i][prb];
	  /* Derivative of LegendreQ at probe */
	  //dleq=DLegendreQ(md,nd,zeta[prb])/LegendreQ(md,nd,zt0);
	  dleq=Tdleq[i][prb];
	  // CC(m,n) coefficient 
	  tcc[i-1]=-Sinh_zt*Sin_et/rr*cos(nd*phi)*cos(md*et)*  \
		   (Sinh_zt/2./wgt*legq+wgt*dleq) \
		   -(1.-Cosh_zt*Cos_et)/rr* \
		   (-md*cos(nd*phi)*sin(md*et)*wgt*legq \
		    +cos(nd*phi)*cos(md*et)*Sin_et/2./wgt*legq);
	  // CS(m,n) coefficient 
	  tcs[i-1]=-Sinh_zt*Sin_et/rr*sin(nd*phi)*cos(md*et)* \
		   (Sinh_zt/2./wgt*legq+wgt*dleq) \
		   -(1.-Cosh_zt*Cos_et)/rr* \
		   (-md*sin(nd*phi)*sin(md*et)*wgt*legq \
		    +sin(nd*phi)*cos(md*et)*Sin_et/2./wgt*legq);
	  // SC(m,n) coefficient 
	  tsc[i-1]=-Sinh_zt*Sin_et/rr*cos(nd*phi)*sin(md*et)* \
		   (Sinh_zt/2./wgt*legq+wgt*dleq) \
		   -(1.-Cosh_zt*Cos_et)/rr* \
		   (md*cos(nd*phi)*cos(md*et)*wgt*legq \
		    +cos(nd*phi)*sin(md*et)*Sin_et/2./wgt*legq);
	  // SS(m,n) coefficient 
	  tss[i-1]=-Sinh_zt*Sin_et/rr*sin(nd*phi)*sin(md*et)* \
		   (Sinh_zt/2./wgt*legq+wgt*dleq) \
		   -(1.-Cosh_zt*Cos_et)/rr* \
		   (md*sin(nd*phi)*cos(md*et)*wgt*legq \
		    +sin(nd*phi)*sin(md*et)*Sin_et/2./wgt*legq);
	} // define tcc loop end

	//Start parallel computation on gpu
	//copy memory
	err = hipMemcpy(d_tcc, tcc, d_size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_tcs, tcs, d_size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_tsc, tsc, d_size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_tss, tss, d_size, hipMemcpyHostToDevice);

	dim3 DimBlock (16,16);
	dim3 DimGrid (dim1/16+1, dim1/16+1);
//	printf("CUDA kernel launch with %d blocks of %d threads\n", DimGrid.x * DimGrid.y, 256);

	vectorProd<<<DimGrid, DimBlock>>>(d_tcc, d_tcs, d_tsc, d_tss, d_M, dim1);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
	  fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}

	for (i=1;i<=dim1;i++){
	  cc[i]+=tcc[i-1]*datatmp;
	  cs[i]+=tcs[i-1]*datatmp;
	  sc[i]+=tsc[i-1]*datatmp;
	  ss[i]+=tss[i-1]*datatmp;
	} // define vector loop end 
	// for each data end
      } // probe loop end 
      //    printf ("\nSlice %d (phi=%lf) done\n", id, phi);
      printf ("\nSlice %d of %d done\n", id, ndata);
    }
  } // id end 

  //copy matrix from device to host
  err = hipMemcpy(h_M, d_M, sizeM, hipMemcpyDeviceToHost);

  //copy matrix back to main
  for (i=0;i<dimM;i++){
    for (j=0;j<dimM;j++) {
      matrix[i][j]=h_M[i*dimM+j];
//      printf("%d,%d,%.17g\n",i,j,matrix[i][j]);
    }
  }
  // Free device global memory
  err = hipFree(d_tcc);
  err = hipFree(d_tcs);
  err = hipFree(d_tsc);
  err = hipFree(d_tss);
  err = hipFree(d_M);

  free(tcc);
  free(tcs);
  free(tsc);
  free(tss);
  for (int il=0;il<=dim1;il++) {
    free(Tlegq[il]);
    free(Tdleq[il]);
  }
  free(Tlegq);
  free(Tdleq);
  free(h_M);
  // Reset the device and exit
  // cudaDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling cudaDeviceReset causes all profile data to be
  // flushed before the application exits
  err = hipDeviceReset();

  //    return 0;
}




